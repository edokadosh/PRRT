#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void generate_uniform_kernel(hiprandState *state,
                                int n,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int count = 0;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int i = 0; i < n; i++) {
        x = hiprand_uniform(&localState);
        /* Check if > .5 */
        if(x > .5) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

int main(int argc, char *argv[])
{
    const unsigned int threadsPerBlock = 64;
    const unsigned int blockCount = 64;
    const unsigned int totalThreads = threadsPerBlock * blockCount;

    unsigned int i;
    unsigned int total;
    hiprandState *devStates;
    unsigned int *devResults, *hostResults;
    int sampleCount = 10000;
    bool doubleSupported = 0;
    int device;
    struct hipDeviceProp_t properties;

    /* check for double precision support */
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipGetDeviceProperties(&properties,device));
    if ( properties.major >= 2 || (properties.major == 1 && properties.minor >= 3) ) {
        doubleSupported = 1;
    }

    /* Allocate space for results on host */
    hostResults = (unsigned int *)calloc(totalThreads, sizeof(int));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, totalThreads *
              sizeof(unsigned int)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, totalThreads *
              sizeof(unsigned int)));

    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devStates, totalThreads *sizeof(hiprandState)));

    /* Setup prng states */
    setup_kernel<<<64, 64>>>(devPHILOXStates);

    /* Generate and use uniform pseudo-random  */
    generate_uniform_kernel<<<64, 64>>>(devStates, sampleCount, devResults);

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, totalThreads *
        sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < totalThreads; i++) {
        total += hostResults[i];
    }
    printf("Fraction of uniforms > 0.5 was %10.13f\n",
        (float)total / (totalThreads * sampleCount * 50.0f));

    /* Cleanup */
    CUDA_CALL(hipFree(devStates));
    
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_example PASSED\n");
    return EXIT_SUCCESS;
}