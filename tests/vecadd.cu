#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

int *a, *b;  // host data
int *c, *c2;  // results

__global__ void vecAdd(int *A,int *B,int *C,int N)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   C[i] = A[i] + B[i]; 
}

void vecAdd_h(int *A1,int *B1, int *C1, int N)
{
   for(int i=0;i<N;i++)
      C1[i] = A1[i] * B1[i];
}

int vecadd_main(int argc,char **argv)
{
   printf("Begin \n");
   int n=100000000;
   int nBytes = n*sizeof(int);
   int block_size, block_no; 
   a = (int *)malloc(nBytes);
   b = (int *)malloc(nBytes);
   c = (int *)malloc(nBytes);
   c2 = (int *)malloc(nBytes);
   int *a_d,*b_d,*c_d;
   block_size=4000;
   block_no = n/block_size;
   dim3 dimBlock(block_size,1,1);
   dim3 dimGrid(block_no,1,1);
   for(int i=0;i<n;i++)
      a[i]=i,b[i]=i;

   printf("Allocating device memory on host..\n");
   hipMalloc((void **)&a_d,n*sizeof(int));
   hipMalloc((void **)&b_d,n*sizeof(int));
   hipMalloc((void **)&c_d,n*sizeof(int));

   printf("Copying to device..\n");
   hipMemcpy(a_d,a,n*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(b_d,b,n*sizeof(int),hipMemcpyHostToDevice);

   clock_t start_d=clock();
   printf("Doing GPU Vector add\n");
   vecAdd<<<block_no,block_size>>>(a_d,b_d,c_d,n);
   hipDeviceSynchronize();
   clock_t end_d = clock();

   clock_t start_h = clock();
   printf("Doing CPU Vector add\n");
   vecAdd_h(a,b,c2,n);
   clock_t end_h = clock();

   double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
   double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;
   hipMemcpy(c,c_d,n*sizeof(int),hipMemcpyDeviceToHost);
   printf("%d %f %f\n",n,time_d,time_h);

   hipFree(a_d);
   hipFree(b_d);
   hipFree(c_d);
   return 0;
}